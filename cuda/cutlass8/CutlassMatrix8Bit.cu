#include "hip/hip_runtime.h"
//
// Created by DmiitriyJarosh on 14.11.2019.
//


#define __CUDA_LIBDEVICE__
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "CutlassMatrix8Bit.h"


// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
#include <chrono>
#include <cmath>


//
// CUTLASS includes needed for single-precision GEMM kernel
//


// Defines cutlass::gemm::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/gemm.h"

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
        int8_t *matrix,
        int ldm,
        int rows,
        int columns) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < rows && j < columns) {
        int offset = i + j * ldm;

        matrix[offset] = 0;
        if (i >= rows - 2 && j < 1) {
            matrix[offset] = 0x0;
        }
        if (i < 1 && j >= columns - 2) {
            matrix[offset] = 0x0;
        }
    }
}

__device__ bool * isChanged;

__global__ void resetChanges(bool * isChangedGlobal) {
    isChanged = isChangedGlobal;
    *isChanged = false;
}

// Kernel for matrix sum and checking if they have changed
__global__ void MatAddKernel(int8_t* A, int8_t* B, int8_t* C, int matDim){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int index = col + row * matDim;
    if (col < matDim && row < matDim) {
        C[index] = A[index] | B[index];
        if (C[index] != B[index]) {
            //printf("@@ ");
            *isChanged = true;
        }
    }
}


#if (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 610))
// Defines cutlass::gemm::SgemmTraits, the structural components for single-precision GEMM
#include "cutlass/gemm/igemm_traits.h"
using namespace cutlass::gemm;

#pragma warning( disable : 4503)

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassIGemm(
        int M,
        int N,
        int K,
        int8_t alpha,
        int8_t const *A,
        int lda,
        int8_t const *B,
        int ldb,
        int8_t beta,
        int8_t *C,
        int ldc) {

    // Define type definition for single-precision CUTLASS GEMM with column-major
    // input matrices and 128x128x8 threadblock tile size.
    //
    // Note, GemmTraits<> is a generic template defined for various general matrix product
    // computations within CUTLASS. It is intended to be maximally flexible, and consequently
    // it contains numerous template arguments.
    //
    // To keep the interface manageable, several helpers are defined for plausible compositions
    // including the following example for single-precision GEMM. Typical values are used as
    // default template arguments. See `cutlass/gemm/gemm_traits.h` for more details.
    //
    typedef cutlass::gemm::IgemmTraitsBoolVector<
            cutlass::MatrixLayout::kColumnMajor,   // layout of A matrix
            cutlass::MatrixLayout::kColumnMajor,   // layout of B matrix
            cutlass::Shape<32, 128, 128>,
            int8_t
    > GemmTraits;

    // Define a CUTLASS GEMM type from a GemmTraits<> instantiation.
    typedef cutlass::gemm::Gemm<GemmTraits> Gemm;

    // Construct and initialize CUTLASS GEMM parameters object.
    //
    // One of CUTLASS's design patterns is to define parameters objects that are constructible
    // in host code and passed to kernels by value. These may include pointers, strides, scalars,
    // and other arguments needed by Gemm and its components.
    //
    // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
    // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
    //
    typename Gemm::Params params;

    int result = params.initialize(
            M,     // GEMM M dimension
            N,     // GEMM N dimension
            K,     // GEMM K dimension
            alpha, // scalar alpha
            A,     // matrix A operand
            lda,
            B,     // matrix B operand
            ldb,
            beta,  // scalar beta
            C,     // source matrix C
            ldc,
            C,     // destination matrix C (may be different memory than source C matrix)
            ldc
    );

    if (result) {
        std::cerr << "Failed to initialize CUTLASS Gemm::Params object." << std::endl;
        return hipErrorInvalidValue;
    }

    // Launch the CUTLASS GEMM kernel.
    Gemm::launch(params);

    // Return any errors associated with the launch or hipSuccess if no error.
    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////



/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(int8_t *matrix, int ldm, int rows, int columns, int8_t * matrix_data = nullptr) {

    if (matrix_data != nullptr) {
        hipMemcpy(matrix, matrix_data, rows * columns * sizeof(int8_t), hipMemcpyHostToDevice);
        return hipGetLastError();
    }
    dim3 block(16, 16);
    dim3 grid(
            (rows + block.x - 1) / block.x,
            (columns + block.y - 1) / block.y
    );

    InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns);

    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(int8_t **matrix, int ldm, int rows, int columns, int8_t * matrix_data = nullptr) {
    hipError_t result;

    size_t sizeof_matrix = sizeof(int8_t) * ldm * columns;

    // Allocate device memory.
    result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to allocate matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Clear the allocation.
    result = hipMemset(*matrix, 0, sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to clear matrix device memory: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Initialize matrix elements
    result = InitializeMatrix(*matrix, ldm, rows, columns, matrix_data);

    if (result != hipSuccess) {
        std::cerr << "Failed to initialize matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
int8_t * CutlassGemmSquare(
        int dim,
        int8_t alpha,
        int8_t beta,
        int8_t * matrixA = nullptr) {
    hipError_t result;

    using namespace std::chrono;
    high_resolution_clock::time_point begin_time = high_resolution_clock::now();

    // Compute leading dimensions for each matrix.
    int lda = dim;
    int ldc = dim;

    // Compute size in bytes of the C matrix.
    size_t sizeof_C = sizeof(int8_t) * ldc * dim;

    // Define pointers to mathigh_resolution_clock::time_point algorithm_begin_time = high_resolution_clock::now();rices in GPU device memory.
    int8_t *A;
    int8_t *C_cutlass;

    //
    // Allocate matrices in GPU device memory with arbitrary seeds.
    //

    result = AllocateMatrix(&A, lda, dim, dim, matrixA);

    if (result !=  hipSuccess) {
        return nullptr;
    }

    result = AllocateMatrix(&C_cutlass, ldc, dim, dim, nullptr);

    if (result != hipSuccess) {
        hipFree(A);
        return nullptr;
    }

    //
    // Launch Main Loop.
    //

    bool isChangedHost = true;
    bool * isChangedGlobal;
    hipMalloc((void**)&isChangedGlobal, sizeof(bool));
    dim3 dimBlock(16, 16);
    dim3 dimGrid((int)ceil((double)dim / dimBlock.x), (int)ceil((double)dim / dimBlock.y));

    int8_t * host_cutlass = (int8_t *)calloc(ldc * dim, sizeof(int8_t));

    high_resolution_clock::time_point algorithm_begin_time = high_resolution_clock::now();
    int i = 0;
    while(isChangedHost) {
        i++;
        result = CutlassIGemm(dim, dim, dim, alpha, A, lda, A, lda, beta, C_cutlass, ldc);

        if (result != hipSuccess) {
            std::cerr << "CUTLASS GEMM kernel failed: "
                      << hipGetErrorString(result) << std::endl;
            hipFree(C_cutlass);
            hipFree(A);
            return nullptr;
        }
        resetChanges<<<1,1>>>(isChangedGlobal);
        MatAddKernel<<<dimGrid, dimBlock>>>(C_cutlass, A, C_cutlass, dim);
        hipMemcpy(&isChangedHost, isChangedGlobal, sizeof(bool), hipMemcpyDeviceToHost);
        result = hipMemcpy(A, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

        if (result != hipSuccess) {
            std::cerr << "Failed to copy CUTLASS Loop results to next iteration: "
                      << hipGetErrorString(result) << std::endl;
            hipFree(C_cutlass);
            hipFree(A);
            return nullptr;
        }
    }

    printf("Amount of multiplications: %d\n", i);
    high_resolution_clock::time_point algorithm_end_time = high_resolution_clock::now();

    result = hipMemcpy(host_cutlass, C_cutlass, sizeof_C, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy CUTLASS GEMM results: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_cutlass);
        hipFree(A);

        return nullptr;
    }

    //
    // Free device memory allocations.
    //

    hipFree(C_cutlass);
    hipFree(A);

    high_resolution_clock::time_point end_time = high_resolution_clock::now();

    milliseconds algorithm_elapsed_secs = duration_cast<milliseconds>(algorithm_end_time - algorithm_begin_time);
    milliseconds elapsed_secs = duration_cast<milliseconds>(end_time - begin_time);

    printf(
            "Algorithm time: %u\nTotal time: %u\n",
            (unsigned int)algorithm_elapsed_secs.count(),
            (unsigned int)elapsed_secs.count()
    );

    return host_cutlass;
}

int8_t ** CutlassMatrix8Bit::MultMatrSquare(unsigned char ** A, int size, unsigned char * grammar_body, unsigned int * grammar_tail, int grammar_size) {
    // Scalars used for linear scaling the result of the matrix product.
    int8_t scalars[2] = { 1, 0 };

    int8_t * matrixA = new int8_t[size * size];
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            int offset = i + j * size;
            matrixA[offset] = A[i][j];
        }
    }

    unsigned char * global_device_grammar_body = nullptr;
    unsigned int * global_device_grammar_tail = nullptr;

    hipError_t result;
    result = hipMalloc((void**)&global_device_grammar_body, grammar_size * sizeof(unsigned char));
    if (result != hipSuccess) {
        std::cerr << "Failed to malloc grammar body: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMalloc((void**)&global_device_grammar_tail, grammar_size * sizeof(unsigned int));
    if (result != hipSuccess) {
        std::cerr << "Failed to malloc gramar tail: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpy(global_device_grammar_body, grammar_body, grammar_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar body to device: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpy(global_device_grammar_tail, grammar_tail, grammar_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar tail to device: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }

    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_body), global_device_grammar_body, grammar_size * sizeof(unsigned char));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar body to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_tail), global_device_grammar_tail, grammar_size * sizeof(unsigned int));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar tail to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_size), &grammar_size, sizeof(int));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar size to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    hipDeviceSynchronize();

    //
    // Run the CUTLASS GEMM test.
    //

    int8_t * res = CutlassGemmSquare(
            size,
            scalars[0],     // alpha
            scalars[1],     // beta
            matrixA
    );

    int8_t ** output = new int8_t*[size];
    for (int i = 0; i < size; i++) {
        output[i] = new int8_t[size];
    }


    for (unsigned int i = 0; i < size; i++) {
        for (unsigned int j = 0; j < size; j++) {
            output[i][j] = res[i + j * size];
        }
    }

    return output;
}
#endif
