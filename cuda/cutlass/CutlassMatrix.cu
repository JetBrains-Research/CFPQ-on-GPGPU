#include "hip/hip_runtime.h"
//
// Created by DmiitriiJarosh on 24.09.2019.
//

#define __CUDA_LIBDEVICE__
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "CutlassMatrix.h"


// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
#include <chrono>
#include <cmath>


//
// CUTLASS includes needed for single-precision GEMM kernel
//


// Defines cutlass::gemm::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/gemm.h"

// Defines cutlass::gemm::SgemmTraits, the structural components for single-precision GEMM
#include "cutlass/gemm/sgemm_traits.h"
using namespace cutlass::gemm;


#pragma warning( disable : 4503)

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSGemmNN(
        int M,
        int N,
        int K,
        unsigned int alpha,
        unsigned int const *A,
        int lda,
        unsigned int const *B,
        int ldb,
        unsigned int beta,
        unsigned int *C,
        int ldc) {

    // Define type definition for single-precision CUTLASS GEMM with column-major
    // input matrices and 128x128x8 threadblock tile size.
    //
    // Note, GemmTraits<> is a generic template defined for various general matrix product
    // computations within CUTLASS. It is intended to be maximally flexible, and consequently
    // it contains numerous template arguments.
    //
    // To keep the interface manageable, several helpers are defined for plausible compositions
    // including the following example for single-precision GEMM. Typical values are used as
    // default template arguments. See `cutlass/gemm/gemm_traits.h` for more details.
    //
    typedef cutlass::gemm::SgemmTraits<
            cutlass::MatrixLayout::kColumnMajor,   // layout of A matrix
            cutlass::MatrixLayout::kColumnMajor,   // layout of B matrix
            cutlass::Shape<8, 128, 128>,           // threadblock tile size
            cutlass::gemm::LinearScaling<unsigned int>,
            cutlass::Shape<8, 8, 8>,
            1,
            1,
            int,
            cutlass::gemm::IgemmConfig<cutlass::Shape<8, 128, 128>, cutlass::Shape<8, 8, 8>, 1, 1, false>
    >
            GemmTraits;

    // Define a CUTLASS GEMM type from a GemmTraits<> instantiation.
    typedef cutlass::gemm::Gemm<GemmTraits> Gemm;

    // Construct and initialize CUTLASS GEMM parameters object.
    //
    // One of CUTLASS's design patterns is to define parameters objects that are constructible
    // in host code and passed to kernels by value. These may include pointers, strides, scalars,
    // and other arguments needed by Gemm and its components.
    //
    // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
    // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
    //
    typename Gemm::Params params;

    int result = params.initialize(
            M,     // GEMM M dimension
            N,     // GEMM N dimension
            K,     // GEMM K dimension
            alpha, // scalar alpha
            A,     // matrix A operand
            lda,
            B,     // matrix B operand
            ldb,
            beta,  // scalar beta
            C,     // source matrix C
            ldc,
            C,     // destination matrix C (may be different memory than source C matrix)
            ldc
    );

    if (result) {
        std::cerr << "Failed to initialize CUTLASS Gemm::Params object." << std::endl;
        return hipErrorInvalidValue;
    }

    // Launch the CUTLASS GEMM kernel.
    Gemm::launch(params);

    // Return any errors associated with the launch or hipSuccess if no error.
    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
        unsigned int *matrix,
        int ldm,
        int rows,
        int columns,
        int seed = 0) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < rows && j < columns) {
        int offset = i + j * ldm;

        matrix[offset] = 0;
        if (i >= rows - 2 && j < 1) {
            matrix[offset] = 0x10;
        }
        if (i < 1 && j >= columns - 2) {
            matrix[offset] = 0x01;
        }
    }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(unsigned int *matrix, int ldm, int rows, int columns, unsigned int * matrix_data = nullptr) {

    if (matrix_data != nullptr) {
        hipMemcpy(matrix, matrix_data, rows * columns * sizeof(unsigned int), hipMemcpyHostToDevice);
        return hipGetLastError();
    }
    dim3 block(16, 16);
    dim3 grid(
            (rows + block.x - 1) / block.x,
            (columns + block.y - 1) / block.y
    );

    InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns);

    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(unsigned int **matrix, int ldm, int rows, int columns, unsigned int * matrix_data = nullptr) {
    hipError_t result;

    size_t sizeof_matrix = sizeof(unsigned int) * ldm * columns;

    // Allocate device memory.
    result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to allocate matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Clear the allocation.
    result = hipMemset(*matrix, 0, sizeof_matrix);

    if (result != hipSuccess) {
        std::cerr << "Failed to clear matrix device memory: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    // Initialize matrix elements
    result = InitializeMatrix(*matrix, ldm, rows, columns, matrix_data);

    if (result != hipSuccess) {
        std::cerr << "Failed to initialize matrix: "
                  << hipGetErrorString(result) << std::endl;
        return result;
    }

    return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

__device__ bool * isChanged;

__global__ void resetChanges(bool * isChangedGlobal) {
    isChanged = isChangedGlobal;
    *isChanged = false;
}

// Kernel for matrix sum and checking if they have changed
__global__ void MatAddKernel(unsigned int* A, unsigned int* B, unsigned int* C, int matDim){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int index = col + row * matDim;
    if (col < matDim && row < matDim) {
        C[index] = A[index] | B[index];
        if (C[index] != B[index]) {
            *isChanged = true;
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
unsigned int * CutlassGemmSquare(
        int dim,
        unsigned int alpha,
        unsigned int beta,
        unsigned int * matrixA = nullptr) {
    hipError_t result;

    using namespace std::chrono;
    high_resolution_clock::time_point begin_time = high_resolution_clock::now();

    // Compute leading dimensions for each matrix.
    int lda = dim;
    int ldc = dim;

    // Compute size in bytes of the C matrix.
    size_t sizeof_C = sizeof(unsigned int) * ldc * dim;

    // Define pointers to mathigh_resolution_clock::time_point algorithm_begin_time = high_resolution_clock::now();rices in GPU device memory.
    unsigned int *A;
    unsigned int *C_cutlass;

    //
    // Allocate matrices in GPU device memory with arbitrary seeds.
    //

    result = AllocateMatrix(&A, lda, dim, dim, matrixA);

    if (result !=  hipSuccess) {
        return nullptr;
    }

    result = AllocateMatrix(&C_cutlass, ldc, dim, dim, nullptr);

    if (result != hipSuccess) {
        hipFree(A);
        return nullptr;
    }

    //
    // Launch Main Loop.
    //

    bool isChangedHost = true;
    bool * isChangedGlobal;
    hipMalloc((void**)&isChangedGlobal, sizeof(bool));
    dim3 dimBlock(16, 16);
    dim3 dimGrid((int)ceil((double)dim / dimBlock.x), (int)ceil((double)dim / dimBlock.y));

    unsigned int * host_cutlass = (unsigned int *)calloc(ldc * dim, sizeof(unsigned int));
//    unsigned int * A_r = (unsigned int *)calloc(lda * dim, sizeof(unsigned int));

    high_resolution_clock::time_point algorithm_begin_time = high_resolution_clock::now();

    int i = 0;
    
    while(isChangedHost) {
        i++;
        result = CutlassSGemmNN(dim, dim, dim, alpha, A, lda, A, lda, beta, C_cutlass, ldc);

        if (result != hipSuccess) {
            std::cerr << "CUTLASS GEMM kernel failed: "
                      << hipGetErrorString(result) << std::endl;
            hipFree(C_cutlass);
            hipFree(A);
            return nullptr;
        }

        resetChanges<<<1,1>>>(isChangedGlobal);
        MatAddKernel<<<dimGrid, dimBlock>>>(C_cutlass, A, C_cutlass, dim);
        hipMemcpy(&isChangedHost, isChangedGlobal, sizeof(bool), hipMemcpyDeviceToHost);

        result = hipMemcpy(A, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

        if (result != hipSuccess) {
            std::cerr << "Failed to copy CUTLASS Loop results to next iteration: "
                      << hipGetErrorString(result) << std::endl;
            hipFree(C_cutlass);
            hipFree(A);
            return nullptr;
        }
    }
    printf("Amount of multiplications: %d\n", i);

    high_resolution_clock::time_point algorithm_end_time = high_resolution_clock::now();

    result = hipMemcpy(host_cutlass, C_cutlass, sizeof_C, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy CUTLASS GEMM results: "
                  << hipGetErrorString(result) << std::endl;

        hipFree(C_cutlass);
        hipFree(A);

        return nullptr;
    }

    //
    // Free device memory allocations.
    //

    hipFree(C_cutlass);
    hipFree(A);

    high_resolution_clock::time_point end_time = high_resolution_clock::now();

    milliseconds algorithm_elapsed_secs = duration_cast<milliseconds>(algorithm_end_time - algorithm_begin_time);
    milliseconds elapsed_secs = duration_cast<milliseconds>(end_time - begin_time);

    printf(
            "Algorithm time: %u\nTotal time: %u\n",
            (unsigned int)algorithm_elapsed_secs.count(),
            (unsigned int)elapsed_secs.count()
    );

    return host_cutlass;
}

unsigned int ** CutlassMatrix::MultMatrSquare(unsigned int ** A, int size, unsigned int * grammar_body, unsigned long long * grammar_tail, int grammar_size) {
    // Scalars used for linear scaling the result of the matrix product.
    unsigned int scalars[2] = { 1, 0 };

    unsigned int * matrixA = new unsigned int[size * size];
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            int offset = i + j * size;
            matrixA[offset] = A[i][j];
        }
    }

    unsigned int * global_device_grammar_body = nullptr;
    unsigned long long * global_device_grammar_tail = nullptr;

    hipError_t result;
    result = hipMalloc((void**)&global_device_grammar_body, grammar_size * sizeof(unsigned int));
    if (result != hipSuccess) {
        std::cerr << "Failed to malloc grammar body: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMalloc((void**)&global_device_grammar_tail, grammar_size * sizeof(unsigned long long));
    if (result != hipSuccess) {
        std::cerr << "Failed to malloc gramar tail: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpy(global_device_grammar_body, grammar_body, grammar_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar body to device: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpy(global_device_grammar_tail, grammar_tail, grammar_size * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar tail to device: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }

    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_body_32), global_device_grammar_body, grammar_size * sizeof(unsigned int));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar body to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_tail_32), global_device_grammar_tail, grammar_size * sizeof(unsigned long long));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar tail to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    result = hipMemcpyToSymbol(HIP_SYMBOL(device_grammar_size_32), &grammar_size, sizeof(int));
    if (result != hipSuccess) {
        std::cerr << "Failed to copy grammar size to __const__: "
                  << hipGetErrorString(result) << std::endl;
        return nullptr;
    }
    hipDeviceSynchronize();

    //
    // Run the CUTLASS GEMM test.
    //

    unsigned int * res = CutlassGemmSquare(
            size,
            scalars[0],     // alpha
            scalars[1],     // beta
            matrixA
    );

    unsigned int ** output = new unsigned int*[size];
    for (int i = 0; i < size; i++) {
        output[i] = new unsigned int[size];
    }

    for (unsigned int i = 0; i < size; i++) {
        for (unsigned int j = 0; j < size; j++) {
            output[i][j] = res[i + j * size];
        }
    }

    return output;
}
